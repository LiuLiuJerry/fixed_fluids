#include "hip/hip_runtime.h"
/*
  FLUIDS v.3 - SPH Fluid Simulator for CPU and GPU
  Copyright (C) 2012-2013. Rama Hoetzlein, http://fluids3.com

  Attribute-ZLib license (* See additional part 4)

  This software is provided 'as-is', without any express or implied
  warranty. In no event will the authors be held liable for any damages
  arising from the use of this software.

  Permission is granted to anyone to use this software for any purpose,
  including commercial applications, and to alter it and redistribute it
  freely, subject to the following restrictions:

  1. The origin of this software must not be misrepresented; you must not
     claim that you wrote the original software.
  2. Altered source versions must be plainly marked as such, and must not be
     misrepresented as being the original software.
  3. This notice may not be removed or altered from any source distribution.
  4. Any published work based on this code must include public acknowledgement
     of the origin. This includes following when applicable:
	   - Journal/Paper publications. Credited by reference to work in text & citation.
	   - Public presentations. Credited in at least one slide.
	   - Distributed Games/Apps. Credited as single line in game or app credit page.	 
	 Retaining this additional license term is required in derivative works.
	 Acknowledgement may be provided as:
	   Publication version:  
	      2012-2013, Hoetzlein, Rama C. Fluids v.3 - A Large-Scale, Open Source
	 	  Fluid Simulator. Published online at: http://fluids3.com
	   Single line (slides or app credits):
	      GPU Fluids: Rama C. Hoetzlein (Fluids v3 2013)

 Notes on Clause 4:
  The intent of this clause is public attribution for this contribution, not code use restriction. 
  Both commerical and open source projects may redistribute and reuse without code release.
  However, clause #1 of ZLib indicates that "you must not claim that you wrote the original software". 
  Clause #4 makes this more specific by requiring public acknowledgement to be extended to 
  derivative licenses. 

*/

#define CUDA_KERNEL
#include "fluid_system_kern.cuh"

#include "cutil_math.h"

#include "radixsort.cu"						// Build in RadixSort

__constant__ FluidParams		simData;
__constant__ uint				gridActive;

__global__ void insertParticles ( bufList buf, int pnum )
{
	uint i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;	// particle index				
	if ( i >= pnum ) return;

	register float3 gridMin = simData.gridMin; //���е���͵�
	register float3 gridDelta = simData.gridDelta; //���еĴ�С
	register int3 gridRes = simData.gridRes;
	register int3 gridScan = simData.gridScanMax; //ɨ������ֵ
	register float poff = simData.psmoothradius / simData.psimscale;//********???

	register int		gs;
	register float3		gcf;
	register int3		gc;

	gcf = (buf.mpos[i] - gridMin) * gridDelta; //������㵽Ŀǰ����λ�õ�����
	gc = make_int3( int(gcf.x), int(gcf.y), int(gcf.z) ); //�������
	gs = (gc.y * gridRes.z + gc.z)*gridRes.x + gc.x; //�����������е����
	if ( gc.x >= 1 && gc.x <= gridScan.x && gc.y >= 1 && gc.y <= gridScan.y && gc.z >= 1 && gc.z <= gridScan.z ) {
		buf.mgcell[i] = gs;											// Grid cell insert.����������ӳ�䵽��άblocks��
		buf.mgndx[i] = atomicAdd ( &buf.mgridcnt[ gs ], 1 );		// Grid counts.// mgridcnt[gs]+1��mgndx��¼mgridcntԭ����ֵ

		gcf = (-make_float3(poff,poff,poff) + buf.mpos[i] - gridMin) * gridDelta;
		gc = make_int3( int(gcf.x), int(gcf.y), int(gcf.z) );
		gs = ( gc.y * gridRes.z + gc.z)*gridRes.x + gc.x;		
	} else {
		buf.mgcell[i] = GRID_UNDEF;		
	}
}

// the mutex variable
__device__ int g_mutex = 0;

// GPU simple synchronization function
__device__ void __gpu_sync(int goalVal)
{

	__threadfence ();

	// only thread 0 is used for synchronization
	if (threadIdx.x == 0) 
		atomicAdd(&g_mutex, 1);
	
	// only when all blocks add 1 to g_mutex will
	// g_mutex equal to goalVal
	while(g_mutex < goalVal) {			// infinite loop until g_mutx = goalVal
	}

	if ( blockIdx.x == 0 && threadIdx.x == 0 ) g_mutex = 0;
	
	__syncthreads();
}

// countingSortInPlace -- GPU_SYNC DOES NOT WORK
/*uint i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;		// particle index				
	if ( i >= pnum ) { __gpu_sync ( 2 ); return; }

	register float3	ipos, ivel, iveleval, iforce;
	register float	ipress, idens;
	register int	icell, indx, iclr;

	icell = buf.mgcell [ i ];
	indx = buf.mgndx [ i ];
	int sort_ndx = buf.mgridoff[ icell ] + indx;				// global_ndx = grid_cell_offet + particle_offset
	if ( icell == GRID_UNDEF ) { __gpu_sync ( 2 ); return; }

	ipos = buf.mpos [ i ];
	ivel = buf.mvel [ i ];
	iveleval = buf.mveleval [ i ];
	iforce = buf.mforce [ i ];
	ipress = buf.mpress [ i ];
	idens = buf.mdensity [ i ];
	iclr = buf.mclr [ i ];

	__gpu_sync ( 2 ) ; //threadfence();			// make sure every thread in all blocks has their data

	
	buf.mpos [ sort_ndx ] = ipos;
	buf.mvel [ sort_ndx ] = ivel;
	buf.mveleval [ sort_ndx ] = iveleval;
	buf.mforce [ sort_ndx ] = iforce;
	buf.mpress [ sort_ndx ] = ipress;
	buf.mdensity [ sort_ndx ] = idens;
	buf.mclr [ sort_ndx ] = iclr;*/



// Counting Sort - Index
__global__ void countingSortIndex ( bufList buf, int pnum )
{
	uint i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;		// particle index				
	if ( i >= pnum ) return;

	uint icell = buf.mgcell[i];
	uint indx =  buf.mgndx[i];
	int sort_ndx = buf.mgridoff[ icell ] + indx;				// global_ndx = grid_cell_offet + particle_offset
	if ( icell != GRID_UNDEF ) {
		buf.mgrid[ sort_ndx ] = i;					// index sort, grid refers to original particle order
	}
}

// Counting Sort - Full (deep copy)
__global__ void countingSortFull ( bufList buf, int pnum )
{
	uint i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;		// particle index	mul24������������߰�λ����Ϊ�������������̫��Ӧ���Ǽӿ��ٶȰ�			
	if ( i >= pnum ) return; //��ֹԽ��

	// Copy particle from original, unsorted buffer (msortbuf),
	// into sorted memory location on device (mpos/mvel)
	uint icell = *(uint*) (buf.msortbuf + pnum*BUF_GCELL + i*sizeof(uint) );
	uint indx =  *(uint*) (buf.msortbuf + pnum*BUF_GNDX + i*sizeof(uint) );		

	if ( icell != GRID_UNDEF ) {	  
		// Determine the sort_ndx, location of the particle after sort
	    int sort_ndx = buf.mgridoff[ icell ] + indx;				// global_ndx = grid_cell_offet + particle_offset	
		
		// Find the original particle data, offset into unsorted buffer (msortbuf)
		char* bpos = buf.msortbuf + i*sizeof(float3);

		// Transfer data to sort location
		buf.mgrid[ sort_ndx ] = sort_ndx;			// full sort, grid indexing becomes identity		
		buf.mpos[ sort_ndx ] =		*(float3*) (bpos);
		buf.mvel[ sort_ndx ] =		*(float3*) (bpos + pnum*BUF_VEL );
		buf.mveleval[ sort_ndx ] =	*(float3*) (bpos + pnum*BUF_VELEVAL );
		buf.mforce[ sort_ndx ] =	*(float3*) (bpos + pnum*BUF_FORCE );
		buf.mpress[ sort_ndx ] =	*(float*) (buf.msortbuf + pnum*BUF_PRESS + i*sizeof(float) );
		buf.mdensity[ sort_ndx ] =	*(float*) (buf.msortbuf + pnum*BUF_DENS + i*sizeof(float) );
		buf.mclr[ sort_ndx ] =		*(uint*) (buf.msortbuf + pnum*BUF_CLR+ i*sizeof(uint) );		// ((uint) 255)<<24; -- dark matter
		buf.mgcell[ sort_ndx ] =	icell;
		buf.mgndx[ sort_ndx ] =		indx;		
	}
}

// ***** UNUSED CODE (not working) ******
__global__ void countActiveCells ( bufList buf, int pnum )
{	
	if ( threadIdx.x == 0 ) {		
		// use only one processor
		
		//gridActive = -1;

		int last_ndx = buf.mgridoff [ simData.gridTotal-1 ] + buf.mgridcnt[ simData.gridTotal-1 ] - 1;
		int last_p = buf.mgrid[ last_ndx ];
		int last_cell = buf.mgcell[ last_p ];
		int first_p = buf.mgrid[ 0 ];
		int first_cell = buf.mgcell[ first_p ] ;

		int cell, cnt = 0, curr = 0;
		cell = first_cell;
		while ( cell < last_cell ) {			
			buf.mgridactive[ cnt ] = cell;			// add cell to active list
			cnt++;
			curr += buf.mgridcnt[cell];				// advance to next active cell
			// id = buf.mgrid[curr];				// get particle id -- when unsorted only
			cell = buf.mgcell [ curr ];				// get cell we are in -- use id when unsorted
		}
		// gridActive = cnt;
	}
	__syncthreads();
}


__device__ float contributePressure ( int i, float3 p, int cell, bufList buf )
{			
	float3 dist;
	float dsq, c, sum;
	register float d2 = simData.psimscale * simData.psimscale;
	register float r2 = simData.r2 / d2;
	
	sum = 0.0;

	if ( buf.mgridcnt[cell] == 0 ) return 0.0;
	
	int cfirst = buf.mgridoff[ cell ];
	int clast = cfirst + buf.mgridcnt[ cell ];
	
	for ( int cndx = cfirst; cndx < clast; cndx++ ) {
		dist = p - buf.mpos[ buf.mgrid[cndx] ];
		dsq = (dist.x*dist.x + dist.y*dist.y + dist.z*dist.z);
		if ( dsq < r2 && dsq > 0.0) {  //�ж����ӵľ���
			c = (r2 - dsq)*d2;
			sum += c * c * c;				
		} 
	}
	
	return sum;
}
			
__global__ void computePressure ( bufList buf, int pnum )
{
	uint i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;	// particle index				
	if ( i >= pnum ) return;

	// Get search cell
	int nadj = (1*simData.gridRes.z + 1)*simData.gridRes.x + 1;
	uint gc = buf.mgcell[ i ];  //��i���������ڵ�����
	if ( gc == GRID_UNDEF ) return;						// particle out-of-range
	gc -= nadj;

	// Sum Pressures
	float3 pos = buf.mpos[ i ];
	float sum = 0.0;
	for (int c=0; c < simData.gridAdjCnt; c++) {//��Χ��gridAdjCnt������
		sum += contributePressure ( i, pos, gc + simData.gridAdj[c], buf );//��c������ĵ�ַ
	}
	__syncthreads();
		
	// Compute Density & Pressure
	sum = sum * simData.pmass * simData.poly6kern; //�˴��õ������ܶ�
	if ( sum == 0.0 ) sum = 1.0;
	buf.mpress[ i ] = ( sum - simData.prest_dens ) * simData.pintstiff;
	buf.mdensity[ i ] = 1.0f / sum;
}

		
__global__ void computeQuery ( bufList buf, int pnum )
{
	uint i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;	// particle index				
	if ( i >= pnum ) return;

	// Get search cell
	int nadj = (1*simData.gridRes.z + 1)*simData.gridRes.x + 1;
	uint gc = buf.mgcell[ i ];
	if ( gc == GRID_UNDEF ) return;						// particle out-of-range
	gc -= nadj;

	// Sum Pressures
	float3 pos = buf.mpos[ i ];
	float sum = 0.0;
	for (int c=0; c < simData.gridAdjCnt; c++) {
		sum += 1.0;
	}
	__syncthreads();
	
}

/*FindNeighbors
int cid = blockIdx.x * blockSize.x + blockIdx.y;   // cluster id	
int pid = threadIdx.x;		           // 0 to 85 (max particles per cell)	
__shared__ Particle  clist[ 85 ];	
__shared__ Particle  plist[ 85*8 ];
if ( pid < clusterCnt[cid] )  
	clist [ pid ] = particles [ clusterNdx[cid] + pid ];

for ( gid = 0;  gid < 8;  gid++ ) {
	if ( pid < gridCnt[  cid + group[gid] ] )  
		plist [ cid*CELL_CNT + pid ] = particles [ sortNdx[ cid + group[gid] ]  + pid ]; 	}

__syncthreads();	
	
for ( int j = 0; j < cellcnt;  j++ ) {
	dst = plist[ pid ] - plist[ j ];
	if ( dst < R2 ) {
     		  ...
	}
}*/

/*grid		    block
<gx, gy, gz>    <1, 32, 64>
256, 256, 256  
total:  */


#define LOCAL_PMAX		896
#define NUM_CELL		27
#define LAST_CELL		26
#define CENTER_CELL		13

__global__ void computePressureGroup ( bufList buf, int pnum )
{
	__shared__ float3	cpos[ LOCAL_PMAX ];

	__shared__ int		ncnt[ NUM_CELL ];
	__shared__ int		ngridoff[ NUM_CELL ];
	__shared__ int		noff[ NUM_CELL ];
	
	int bid = __mul24( blockIdx.y, gridDim.x ) + blockIdx.x;
	if ( bid > gridActive ) return;				// block must be in a valid grid
	uint cell = buf.mgridactive [ bid ];		// get grid cell (from blockID 1:1)
	register int i = -1;
	register float3 ipos;

	uint ndx = threadIdx.x;							
	if ( ndx < buf.mgridcnt[cell] ) {
		i = buf.mgridoff[cell] + ndx;		// particle id to process
		ipos = buf.mpos[ i ];
	}
	int gid = threadIdx.x;

	register float d2 = simData.psimscale * simData.psimscale;
	register float r2 = simData.r2 / d2;
	register float3 dist;
	register float c, dsq, sum;
	int neighbor;

	// copy neighbor cell counts to shared mem
	if ( gid < NUM_CELL ) {
		int nadj = (1*simData.gridRes.z + 1)*simData.gridRes.x + 1;
		neighbor = cell - nadj + simData.gridAdj[gid];					// neighbor cell id
		ncnt[gid] = buf.mgridcnt [ neighbor ];	
		ngridoff[gid] = buf.mgridoff [ neighbor ];
	}
	__syncthreads ();

	if ( gid == 0 ) {									// compute neighbor local ndx (as prefix sum)
		int nsum = 0;
		for (int z=0; z < NUM_CELL; z++) {				// 27-step prefix sum
			noff[z] = nsum;
			nsum += ncnt[z];
		}
	}
	__syncthreads ();

	// copy particles into shared memory
	if ( gid < NUM_CELL ) {
		for (int j=0; j < ncnt[gid]; j++ ) {
			neighbor = buf.mgrid [ ngridoff[gid] + j ];		// neighbor particle id
			ndx = noff[ gid ] + j;
			cpos[ ndx ] = buf.mpos [ neighbor ];
		}
	}
	__syncthreads ();

	
	// compute pressure for current particle
	if ( i == -1 ) return;
	
	int jnum = noff[LAST_CELL] + ncnt[LAST_CELL];
	sum = 0.0;
	for (int j = 0; j < jnum; j++) {
		dist = ipos - cpos[ j ];
		dsq = (dist.x*dist.x + dist.y*dist.y + dist.z*dist.z);			
		if ( dsq > 0.0 && dsq < r2 ) {
			c = (r2 - dsq)*d2;
			sum += c * c * c;
		}
	}	
	__syncthreads ();

	// put result into global mem
	sum = sum * simData.pmass * simData.poly6kern;
	if ( sum == 0.0 ) sum = 1.0;
	buf.mpress[ i ] = ( sum - simData.prest_dens ) * simData.pintstiff;
	buf.mdensity[ i ] = 1.0f / sum; 	
}


__device__ float3 contributeForce ( int i, float3 ipos, float3 iveleval, float ipress, float idens, int cell, bufList buf )
{			
	float dsq, c;	
	float pterm;
	float3 dist, force;	
	int j;					

	if ( buf.mgridcnt[cell] == 0 ) return make_float3(0,0,0);	

	force = make_float3(0,0,0);

	for ( int cndx = buf.mgridoff[ cell ]; cndx < buf.mgridoff[ cell ] + buf.mgridcnt[ cell ]; cndx++ ) {										
		j = buf.mgrid[ cndx ];				
		dist = ( ipos - buf.mpos[ j ] );		// dist in cm   ��������
		dsq = (dist.x*dist.x + dist.y*dist.y + dist.z*dist.z);
		if ( dsq < simData.rd2 && dsq > 0) {			
			dsq = sqrt(dsq * simData.d2);
			c = ( simData.psmoothradius - dsq ); 
			pterm = simData.psimscale * -0.5f * c * simData.spikykern * ( ipress + buf.mpress[ j ] ) / dsq;		
			//�Ͳ�����m�ˣ���������ٶȻ�Ҫ������...���ǳ���idens�Ǹ���ѽ����Ȼ�ǵ����Ļ�
			force += ( pterm * dist + simData.vterm * ( buf.mveleval[ j ] - iveleval )) * c * idens * (buf.mdensity[ j ] );
		}	
	}
	return force;
}


__global__ void computeForce ( bufList buf, int pnum)
{			
	uint i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;	// particle index				
	if ( i >= pnum ) return;

	// Get search cell	
	uint gc = buf.mgcell[ i ];
	if ( gc == GRID_UNDEF ) return;						// particle out-of-range
	gc -= (1*simData.gridRes.z + 1)*simData.gridRes.x + 1;

	// Sum Pressures	
	register float3 force;
	force = make_float3(0,0,0);		

	for (int c=0; c < simData.gridAdjCnt; c++) {
		force += contributeForce ( i, buf.mpos[ i ], buf.mveleval[ i ], buf.mpress[ i ], buf.mdensity[ i ], gc + simData.gridAdj[c], buf );
	}
	buf.mforce[ i ] = force;
}
	

/*__global__ void computeForceNbr ( char* bufPnts, int* bufGrid, int numPnt )
{		
	uint ndx = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;	// particle index		
	if ( ndx >= numPnt ) return;
				
	char* ioffs = bufPnts + __mul24(ndx, simData.stride );
	float3 ipos = *(float3*)	(ioffs + OFFSET_POS);
	float3 ivelval = *(float3*)	(ioffs + OFFSET_VELEVAL);
	float press = *(float*)		(ioffs + OFFSET_PRESS);
	float dens =  *(float*)		(ioffs + OFFSET_DENS);
	int icnt =  *(int*)			(ioffs + OFFSET_NBRCNT);

	char* joffs;
	float3 jpos, jveleval;

	float3 dist, force;		
	float c, ndistj, pterm, dterm, vterm;
		
	vterm = simData.lapkern * simData.visc;
		
	force = make_float3(0,0,0);
	for (int nbr=0; nbr < icnt; nbr++) {		// base 1, n[0] = count
		ndistj = bufNdist[ndx][nbr];
		joffs = bufPnts + __mul24(bufNeighbor[ndx][nbr], simData.stride);
		jpos = *(float3*)		(joffs + OFFSET_POS);
		jveleval = *(float3*)	(joffs + OFFSET_VELEVAL);
		c = ( simData.smooth_rad - ndistj ); 
		dist.x = ( ipos.x - jpos.x );		// dist in cm
		dist.y = ( ipos.y - jpos.y );
		dist.z = ( ipos.z - jpos.z );			
		pterm = simData.sim_scale * -0.5f * c * simData.spikykern * ( press + *(float*)(joffs+OFFSET_PRESS) ) / ndistj;
		dterm = c * dens * *(float*)(joffs+OFFSET_DENS);	
		force.x += ( pterm * dist.x + vterm * ( jveleval.x - ivelval.x )) * dterm;
		force.y += ( pterm * dist.y + vterm * ( jveleval.y - ivelval.y )) * dterm;
		force.z += ( pterm * dist.z + vterm * ( jveleval.z - ivelval.z )) * dterm;			
	}
	*(float3*) ( ioffs + OFFSET_FORCE ) = force;		
}*/

		
__global__ void advanceParticles ( float time, float dt, float ss, bufList buf, int numPnts )
{		
	uint i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;	// particle index				
	if ( i >= numPnts ) return;
	
	if ( buf.mgcell[i] == GRID_UNDEF ) {
		buf.mpos[i] = make_float3(-1000,-1000,-1000);
		buf.mvel[i] = make_float3(0,0,0);
		return;
	}
			
	// Get particle vars
	register float3 accel, norm;
	register float diff, adj, speed;
	register float3 pos = buf.mpos[i];
	register float3 veval = buf.mveleval[i];

	// Leapfrog integration						
	accel = buf.mforce[i]; //ֻ�����force��
	accel *= simData.pmass; //������ʽ��Ӧ�ó�������mass��֮ǰû�ˣ���γ���һ�����Ͳ��ó���ȥ��
		
	// Boundaries
	// Y-axis
	//������һ������б������Ҫ����ͱ߽������б����һ��
	diff = simData.pradius - (pos.y - (simData.pboundmin.y + (pos.x-simData.pboundmin.x)*simData.pground_slope )) * ss;
	if ( diff > EPSILON ) {
		norm = make_float3( -simData.pground_slope, 1.0 - simData.pground_slope, 0);
		adj = simData.pextstiff * diff - simData.pdamp * dot(norm, veval );
		norm *= adj; accel += norm;
	}

	diff = simData.pradius - ( simData.pboundmax.y - pos.y )*ss;
	if ( diff > EPSILON ) {
		norm = make_float3(0, -1, 0); //�ܵ�һ����������
		adj = simData.pextstiff * diff - simData.pdamp * dot(norm, veval ); //***************������
		norm *= adj; accel += norm;//�������ٶ�
	}

	// X-axis    ���Ĵ�С�����Һ���
	diff = simData.pradius - (pos.x - (simData.pboundmin.x + (sin(time*simData.pforce_freq)+1)*0.5 * simData.pforce_min))*ss; //X����ǰ�ܵ�һ��������������
	if ( diff > EPSILON ) {
		norm = make_float3( 1, 0, 0);
		adj = (simData.pforce_min+1) * simData.pextstiff * diff - simData.pdamp * dot(norm, veval );
		norm *= adj; accel += norm;
	}
	diff = simData.pradius - ( (simData.pboundmax.x - (sin(time*simData.pforce_freq)+1)*0.5*simData.pforce_max) - pos.x)*ss;
	if ( diff > EPSILON ) {
		norm = make_float3(-1, 0, 0);
		adj = (simData.pforce_max+1) * simData.pextstiff * diff - simData.pdamp * dot(norm, veval );
		norm *= adj; accel += norm;
	}

	// Z-axis
	diff = simData.pradius - (pos.z - simData.pboundmin.z ) * ss; //z������±߽�
	if ( diff > EPSILON ) {
		norm = make_float3( 0, 0, 1 );
		adj = simData.pextstiff * diff - simData.pdamp * dot(norm, veval );
		norm *= adj; accel += norm;
	}
	diff = simData.pradius - ( simData.pboundmax.z - pos.z )*ss;
	if ( diff > EPSILON ) {
		norm = make_float3( 0, 0, -1 );
		adj = simData.pextstiff * diff - simData.pdamp * dot(norm, veval );
		norm *= adj; accel += norm;
	}
		
	// Gravity******************************************
	accel += simData.pgravity;

	// Accel Limit
	speed = accel.x*accel.x + accel.y*accel.y + accel.z*accel.z;
	if ( speed > simData.AL2 ) {
		accel *= simData.AL / sqrt(speed);
	}

	// Velocity Limit
	float3 vel = buf.mvel[i];
	speed = vel.x*vel.x + vel.y*vel.y + vel.z*vel.z;
	if ( speed > simData.VL2 ) {
		speed = simData.VL2;
		vel *= simData.VL / sqrt(speed);
	}

	// Ocean colors
	if ( speed > simData.VL2*0.2) {
		adj = simData.VL2*0.2;
		buf.mclr[i] += ((  buf.mclr[i] & 0xFF) < 0xFD ) ? +0x00000002 : 0;		// decrement R by one
		buf.mclr[i] += (( (buf.mclr[i]>>8) & 0xFF) < 0xFD ) ? +0x00000200 : 0;	// decrement G by one
		buf.mclr[i] += (( (buf.mclr[i]>>16) & 0xFF) < 0xFD ) ? +0x00020000 : 0;	// decrement G by one
	}
	if ( speed < 0.03 ) {		
		int v = int(speed/.01)+1;
		buf.mclr[i] += ((  buf.mclr[i] & 0xFF) > 0x80 ) ? -0x00000001 * v : 0;		// decrement R by one
		buf.mclr[i] += (( (buf.mclr[i]>>8) & 0xFF) > 0x80 ) ? -0x00000100 * v : 0;	// decrement G by one
	}
	////My Add*************************************************************************************************
	register int phase = buf.mphase[i];
	if (phase < 1){
		buf.mclr[i] = 0xFBFBFBFB;
	}
	////*******************************************************************************************************
	//
	////-- surface particle density 
	//buf.mclr[i] = buf.mclr[i] & 0x00FFFFFF;
	//if ( buf.mdensity[i] > 0.0014 ) buf.mclr[i] += 0xAA000000;

	// Leap-frog Integration
	float3 vnext = accel*dt + vel;				// v(t+1/2) = v(t-1/2) + a(t) dt	
	//My add  **************************************************************************************************
	//buf.mveleval[i] = (vel + vnext) * 0.5;		// v(t+1) = [v(t-1/2) + v(t+1/2)] * 0.5			
	//buf.mvel[i] = vnext;
	
	buf.mveleval[i] = (vel + vnext) * 0.5 * phase;		// v(t+1) = [v(t-1/2) + v(t+1/2)] * 0.5			
	buf.mvel[i] = vnext * phase;
	//**********************************************************************************************************
	buf.mpos[i] += vnext * (dt/ss);						// p(t+1) = p(t) + v(t+1/2) dt		
}


void updateSimParams ( FluidParams* cpufp )
{
	hipError_t status;
	#ifdef CUDA_42
		// Only for CUDA 4.x or earlier. Depricated in CUDA 5.0+
		// Original worked even if symbol was declared __device__
		status = hipMemcpyToSymbol(HIP_SYMBOL( "simData"), cpufp, sizeof(FluidParams) );
	#else
		// CUDA 5.x+. Only works if symbol is declared __constant__
		status = hipMemcpyToSymbol(HIP_SYMBOL( simData), cpufp, sizeof(FluidParams) );
	#endif

	/*app_printf ( "SIM PARAMETERS:\n" );
	app_printf ( "  CPU: %p\n", cpufp );	
	app_printf ( "  GPU: %p\n", &simData );	 */
}


//My add
__device__ int searchIce(int i, float3 ipos, float idens, int cell, bufList buf)
{

	if ( buf.mgridcnt[cell] == 0 ) return 0;	

	int n = 0,j;
	float3 dist;
	float dsq;


	for ( int cndx = buf.mgridoff[ cell ]; cndx < buf.mgridoff[ cell ] + buf.mgridcnt[ cell ]; cndx++ ) {	
		j = buf.mgrid[ cndx ];				
		dist =  ipos - buf.mpos[ j ] ;		// dist in cm   ��������
		dsq = (dist.x*dist.x + dist.y*dist.y + dist.z*dist.z);
		if ( dsq < simData.rd2 && dsq > 0) {
			if(buf.mphase[j] < 0.5)
				n++;
		}
	}

	return n;
}

__global__ void updatePhase (bufList buf, int pnumint, float ss)
{
	uint i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;	// particle index				
	if ( i >= pnumint ) return;

	register int newPhase = 1, n = 0;
	register float diff;
	register float3 pos = buf.mpos[i];

	uint gc = buf.mgcell[ i ];
	if ( gc == GRID_UNDEF ) return;						// particle out-of-range
	gc -= (1*simData.gridRes.z + 1)*simData.gridRes.x + 1;

	for (int c=0; c < simData.gridAdjCnt; c++) {
		n += searchIce( i, buf.mpos[ i ], buf.mdensity[ i ], gc + simData.gridAdj[c], buf);
	}

	if(n > 15)
	{
		newPhase = 0;
	}

	//diff < 0 ˵����û���ף���֮˵��������
	diff = simData.pradius - (pos.y - (simData.pboundmin.y + (pos.x-simData.pboundmin.x)*simData.pground_slope )) * ss;

	if(diff - EPSILON > 0)
	{
		newPhase = 0;
	}

	buf.mphase[i] = newPhase;
}

__device__ float3 contributeTension ( int i, float3 ipos, float idens, int cell, bufList buf)
{
	float dsq, c;	
	float3 dist, tension = make_float3(0,0,0);	
	int j;	

	if ( buf.mgridcnt[cell] == 0 ) return make_float3(0,0,0);	

	tension = make_float3(0,0,0);

	for ( int cndx = buf.mgridoff[ cell ]; cndx < buf.mgridoff[ cell ] + buf.mgridcnt[ cell ]; cndx++ ) {										
		j = buf.mgrid[ cndx ];				
		dist =  ipos - buf.mpos[ j ] ;		// dist in cm   ��������
		dsq = (dist.x*dist.x + dist.y*dist.y + dist.z*dist.z);
		if ( dsq < simData.rd2 && dsq > 0) {			
			dsq = dsq * simData.d2;//��ʵ�ʴ�С
			c = simData.r2 - dsq ;
			float gradient = c * -6 * simData.poly6kern * simData.r2 * simData.r2;
			float3 dn = simData.psimscale * idens * (buf.mdensity[ j ])*  gradient * dist * (1.0/sqrt(dsq));	
			tension  += dn;
			//buf.msurfaceTension[i] += dn ;
			//buf.msurfaceTension[j] -= 0.5 * dn ;

		}	
	}

	return tension;

}

//__device__ float contributeWeight (int i, float3 ipos, int cell, bufList buf)
//{
//	float total_Weight = 0,dsq,c;
//	int j;
//	float3 dist;
//	for ( int cndx = buf.mgridoff[ cell ]; cndx < buf.mgridoff[ cell ] + buf.mgridcnt[ cell ]; cndx++ ) {										
//		j = buf.mgrid[ cndx ];				
//		dist =  ipos - buf.mpos[ j ];		// dist in cm   ��������
//		dsq = (dist.x*dist.x + dist.y*dist.y + dist.z*dist.z);
//		if ( dsq < simData.rd2 && dsq > 0) {			
//			c = simData.r2 - dsq * simData.d2 ;
//			total_Weight += c / simData.r2;
//		}
//	}
//	return total_Weight;
//}

__global__ void computeTension(bufList buf, int pnum)
{
	uint i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;	// particle index				
	if ( i >= pnum ) return;

	// Get search cell	
	uint gc = buf.mgcell[ i ];
	if ( gc == GRID_UNDEF ) return;						// particle out-of-range
	gc -= (1*simData.gridRes.z + 1)*simData.gridRes.x + 1;

	// 	
	register float3 tension,smoothedtension,n;
	register float total_weight = 0;

	tension = make_float3(0,0,0);
	smoothedtension = make_float3(0,0,0);
	n = make_float3(0,0,0);

	for (int c=0; c < simData.gridAdjCnt; c++) {
		n += contributeTension ( i, buf.mpos[ i ], buf.mdensity[ i ], gc + simData.gridAdj[c], buf);
		//total_weight += contributeWeight ( i, buf.mpos[ i ], gc + simData.gridAdj[c], buf);
	}
	__syncthreads();

	//��ʱ��tension��ʵ��n   loop 2th
	float magnitude = n.x * n.x + n.y * n.y + n.z * n.z;
	magnitude = sqrt(magnitude);

	if(magnitude > 0)
		tension = magnitude * 30 * n;
	buf.msurfaceTension[i] = tension;
	__syncthreads();

	for (int cc=0; cc < simData.gridAdjCnt; cc++) {
		float weight = 0,dsq,c;
		int j;
		int cell = gc + simData.gridAdj[cc];
		float3 dist,ipos;
		for ( int cndx = buf.mgridoff[ cell ]; cndx < buf.mgridoff[ cell ] + buf.mgridcnt[ cell ]; cndx++ ) {										
			j = buf.mgrid[ cndx ];	
			ipos = buf.mpos[i];
			dist =  ipos - buf.mpos[ j ];		// dist in cm   ��������
			dsq = dist.x*dist.x + dist.y*dist.y + dist.z*dist.z;
			if ( dsq < simData.rd2 && dsq > 0) {			
				c = simData.r2 - dsq * simData.d2 ;
				weight = c / simData.r2;
				total_weight += weight;
				smoothedtension += weight * buf.msurfaceTension[j];
			}
		}
	}

	if(total_weight > 0)
		buf.mforce[ i ] += smoothedtension/total_weight;
}
